#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include <stdio.h>
#include <time.h>
#include <iostream>


#include "Matrix.h"
#include "config.h"
#include "SimpleIm.h"
#include "SharedMatrixTranspose.h"
#include "Func.cu"

using namespace std;


//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	const int WIDTH = 4096 ;
	const int HEIGHT = 4096;
	const int Length = WIDTH*HEIGHT;
	//int array[1048576];
	//Matrix<int, WIDTH, HEIGHT> matrix(array);
	
	float *input = new float[Length];
	for (int i = 0; i < Length; i++)
		input[i] = i;
	const int num_rows = WIDTH;
	const int num_cols = HEIGHT;

	float *input_matrix;
	float *output_matrix;

	hipMalloc((void**)&input_matrix, sizeof(float) * HEIGHT *HEIGHT);
	hipMalloc((void**)&output_matrix, sizeof(float) * HEIGHT *HEIGHT);

	hipMemcpy(input_matrix, input, sizeof(float) * HEIGHT *HEIGHT, hipMemcpyHostToDevice);

	int grid_size = (num_rows - 1) / BLOCKSIZE + 1;
	dim3 gridDim(grid_size, grid_size);
	dim3 blockDim(BLOCKSIZE, BLOCKSIZE);

	clock_t time = clock();

	simpleIMP <<<gridDim, blockDim >>> (input_matrix, output_matrix, num_rows, num_cols);

	clock_t result = clock() - time;
	sharedMatrixTranspose << <gridDim, blockDim >> > (input_matrix, output_matrix, num_rows, num_cols);

	clock_t result2 = clock() - result;

	hipMemcpy(input,output_matrix, sizeof(float) * HEIGHT * HEIGHT, hipMemcpyDeviceToHost);


	//record the execution time 
	float gpu_elapsed_time_ms = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	hipEventRecord(start, 0);
	baseTranspose<float> << <gridDim, blockDim >> > (input_matrix, output_matrix, num_rows, num_cols);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
	
	std::cout << gpu_elapsed_time_ms << std::endl;

	hipEventRecord(start, 0);

	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	coalescedBlockWiseTranspose<float> << <gridDim, blockDim,0,stream1>> > (input_matrix, output_matrix, num_rows, num_cols);
	coalescedBlockWiseTransposeWithNoBankConflicts<float> << <gridDim, blockDim,0,stream2>> > (input_matrix, output_matrix, num_rows, num_cols);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);


	
	std::cout << gpu_elapsed_time_ms << std::endl;

	hipFree(input_matrix);
	hipFree(output_matrix);
//	for (int i = 0; i < 1024; i++)
		//cout<<input[i]<<endl;
	return 0;
}
